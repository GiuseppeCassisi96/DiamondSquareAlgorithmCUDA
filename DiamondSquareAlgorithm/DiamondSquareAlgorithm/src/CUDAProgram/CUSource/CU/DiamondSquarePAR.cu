#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
//for __syncthreads
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__
#include <hip/device_functions.h>
#include <iostream>
#include <cstdlib>
#include "CUDAProgram/API/DSParallel/DiamondSquarePAR.h"
#include "CUDAProgram/API/Utils/RandNumGenDevice.h"
#include "CPUProgram/API/Utils/RandNumberGenerator.h"


// SETTINGS
#define SQUARE_BLOCK_X_SIZE		16  //8 or 16
#define MAX_BLOCK_SIZE			32 //16 or 32

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}



int blockSizeDiamond, blockXSizeSquare, blockYSizeSquare;
int gridSizeDiamond, gridSizeSquare;
RandNumberGenerator generator;

//DEVICE FUNCTIONS
__device__ float clamp(float x, float min_val, float max_val)
{
	return fminf(fmaxf(x, min_val), max_val);
}


__global__ void KERNEL_InitCorners(float* HeightMap, int HeightMapSize, int chunkSize,
	float minHeightValue, float maxHeightValue, hiprandState* ranGenStates)
{
	unsigned const int idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned const int idy = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned const int x = idx * chunkSize;
	unsigned const int y = idy * chunkSize;
	unsigned const int index = y * HeightMapSize + x;
	if(index < HeightMapSize * HeightMapSize)
	{
		float value = GenerateFloatInRangeGPU(ranGenStates, minHeightValue, maxHeightValue, index);
		value = clamp(value, minHeightValue, maxHeightValue);
		HeightMap[index] = value;
	}
}

__global__ void KERNEL_DiamondStep(int chunkSize, float* HeightMap, int HeightMapSize, 
	float randMagnitude, int half, hiprandState* ranGenStates,
	float minHeightValue, float maxHeightValue)
{
	unsigned const int x = (blockDim.x * blockIdx.x + threadIdx.x) * chunkSize;
	unsigned const int y = (blockDim.y * blockIdx.y + threadIdx.y) * chunkSize;
	if (x > HeightMapSize || y > HeightMapSize)
	{
		return;
	}
	float value = HeightMap[y * HeightMapSize + x] + HeightMap[y * HeightMapSize + (x + chunkSize)] +
		HeightMap[(y + chunkSize) * HeightMapSize + x] + 
		HeightMap[(y + chunkSize) * HeightMapSize + (x + chunkSize)];
	value /= 4.0f;
	value += GenerateFloatGPU(ranGenStates, randMagnitude, (blockDim.x * blockIdx.x + threadIdx.x) *
		HeightMapSize + (blockDim.y * blockIdx.y + threadIdx.y));
	value = clamp(value, minHeightValue, maxHeightValue);
	HeightMap[(y + half) * HeightMapSize + (x + half)] = value;
	
}

__global__ void KERNEL_SquareStep(int chunkSize, float* HeightMap, int HeightMapSize,
	float randMagnitude, int half, hiprandState* ranGenStates, float minHeight, float maxHeight)
{
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned const  int x = (blockIdx.x * blockDim.x + threadIdx.x) * chunkSize * (y % 2 == 0)
		+ y * half * (y % 2 != 0);
	y = (y * half + half) * (y % 2 == 0)
		+ (blockIdx.x * blockDim.x + threadIdx.x) * chunkSize * (y % 2 != 0);
	
	if (x > HeightMapSize || y > HeightMapSize)
	{
		return;
	}
	float value = 0.0f;

	//FIRST ELEMENT
	int currentIndex = static_cast<int>((x - half) * HeightMapSize + y);
	if(currentIndex >= 0)
	{
		value += HeightMap[currentIndex];
	}
	else
	{
		value += GenerateFloatInRangeGPU(ranGenStates, minHeight, maxHeight, 
			(blockIdx.y * blockDim.y + threadIdx.y) * HeightMapSize + (blockIdx.x * blockDim.x + threadIdx.x));
	}

	//SECOND ELEMENT
	currentIndex = static_cast<int>(x * HeightMapSize + (y - half));
	if (currentIndex >= 0)
	{
		value += HeightMap[currentIndex];
	}
	else
	{
		value += GenerateFloatInRangeGPU(ranGenStates, minHeight, maxHeight, 
			(blockIdx.y * blockDim.y + threadIdx.y) * HeightMapSize + (blockIdx.x * blockDim.x + threadIdx.x));
	}

	//THIRD ELEMENT
	currentIndex = static_cast<int>(x * HeightMapSize + (y + half));
	if (currentIndex < HeightMapSize * HeightMapSize)
	{
		value += HeightMap[currentIndex];
	}
	else
	{
		value += GenerateFloatInRangeGPU(ranGenStates, minHeight, maxHeight, 
			(blockIdx.y * blockDim.y + threadIdx.y) * HeightMapSize + (blockIdx.x * blockDim.x + threadIdx.x));
	}

	//FOURTH ELEMENT
	currentIndex = static_cast<int>((x + half) * HeightMapSize + y);
	if (currentIndex < HeightMapSize * HeightMapSize)
	{
		value += HeightMap[currentIndex];
	}
	else
	{
		value += GenerateFloatInRangeGPU(ranGenStates, minHeight, maxHeight, 
			(blockIdx.y * blockDim.y + threadIdx.y) * HeightMapSize + (blockIdx.x * blockDim.x + threadIdx.x));
	}

	//FINAL COMPUTATION
	value /= 4.0f;
	value += GenerateFloatGPU(ranGenStates, randMagnitude, 
		(blockIdx.y * blockDim.y + threadIdx.y) * HeightMapSize + (blockIdx.x * blockDim.x + threadIdx.x));
	value = clamp(value, minHeight, maxHeight);
	HeightMap[x * HeightMapSize + y] = value;
}


//HOST FUNCTIONS
void DiamondSquarePAR::ComputeBlockGridSizes()
{
	//It can be 1 or 32 at maximum. The size is 2^k until it reaches the max block size 
	blockSizeDiamond = TwoKElements <= MAX_BLOCK_SIZE ? TwoKElements : MAX_BLOCK_SIZE;
	/* The size is 2^k + 1 until it reaches the max X dim for the block, in this case will be always
	* the max X dim for the block*/
	blockXSizeSquare = TwoKElements < SQUARE_BLOCK_X_SIZE ? blockSizeDiamond + 1 : SQUARE_BLOCK_X_SIZE;
	//The size is 2^(k+1) until it reaches the X dim for the block, in this case the dim will be always 2^k
	blockYSizeSquare = TwoKElements <= SQUARE_BLOCK_X_SIZE ? TwoKElements * 2 : blockSizeDiamond;

	/*With this formula every time my TwoKElements over a multiple of 32 (32, 64, 96, ecc) I increment the
	* grid size exponentially (powers of 2)*/
	gridSizeDiamond = (TwoKElements + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;
	/*With this formula every time the double of TwoKElements over a multiple of 32 (32, 64, 96, ecc)
	 *I increment the grid size exponentially (powers of 2)*/
	gridSizeSquare = (TwoKElements * 2 + MAX_BLOCK_SIZE - 1) / MAX_BLOCK_SIZE;


}


DiamondSquarePAR::DiamondSquarePAR(int NSize, float minHeightValue, float maxHeightValue, float randomValue) : N(NSize), minHeightValue(minHeightValue),
maxHeightValue(maxHeightValue), randomMagnitude(randomValue)
{
	//Height map size computation 
	HeightMapSize = static_cast<int>(std::pow(2, N) + 1);
	totalSize = HeightMapSize * HeightMapSize;

	//Bytesize computation
	byteSize =  static_cast<int>(sizeof(float) * totalSize);
	const int byteSizeForRandom = static_cast<int>(sizeof(hiprandState) * totalSize);

	//CPU memory allocation;
	HeightMap = static_cast<float*>(malloc(byteSize));

	//GPU memory allocation and setting
	hipMalloc((void**)&HeightMapGPU, byteSize);
	hipMalloc((void**)&states, byteSizeForRandom);
	hipMemset(HeightMapGPU, 0.0f, byteSize);

	//chunkSize, half and threadAmount computation 
	chunkSize = HeightMapSize - 1;
	half = chunkSize / 2;
	TwoKElements = (HeightMapSize - 1) / chunkSize;
}

void DiamondSquarePAR::InitializationDS()
{
	//InitCurand kernel threads configuration
	constexpr int xDimBlock = MAX_BLOCK_SIZE;
	constexpr int yDimBlock = MAX_BLOCK_SIZE;
	const int xDimGrid = static_cast<int>(ceil(static_cast<float>(HeightMapSize) / static_cast<float>(xDimBlock)));
	const int yDimGrid = static_cast<int>(ceil(static_cast<float>(HeightMapSize) / static_cast<float>(yDimBlock)));
	dim3 randblock_dim(xDimBlock, yDimBlock);
	dim3 randgrid_dim(xDimGrid, yDimGrid);

	//Rand seed generation
	const int randSeed = static_cast<int>( generator.GenerateFloat(100.0f));

	//InitCurand kernel execution
	InitCurand<<<randgrid_dim, randblock_dim>>> (states, HeightMapSize, randSeed);
	hipDeviceSynchronize();
	cudaCheckError()

	//InitCorners kernel execution
	KERNEL_InitCorners <<<2, 2>>> (HeightMapGPU, HeightMapSize, chunkSize,
		minHeightValue, maxHeightValue, states);
	cudaCheckError()
	hipDeviceSynchronize();
}

void DiamondSquarePAR::DiamondStep()
{
	dim3 block_dim(blockSizeDiamond, blockSizeDiamond);
	dim3 grid_dim(gridSizeDiamond, gridSizeDiamond);
	KERNEL_DiamondStep <<<grid_dim, block_dim >>> (chunkSize, HeightMapGPU,
		HeightMapSize, randomMagnitude, half, states, minHeightValue, maxHeightValue);
	cudaCheckError()
}

void DiamondSquarePAR::SquareStep()
{
	dim3 block_dim(blockXSizeSquare, blockYSizeSquare);
	dim3 grid_dim(gridSizeSquare, gridSizeSquare);
	KERNEL_SquareStep <<<grid_dim, block_dim >>> (chunkSize, HeightMapGPU,
		HeightMapSize, randomMagnitude, half, states, minHeightValue, maxHeightValue);
	cudaCheckError()
}
void DiamondSquarePAR::RunDiamondSquare()
{
	while (chunkSize > 1)
	{
		ComputeBlockGridSizes();
		DiamondStep();
		hipDeviceSynchronize();
		SquareStep();
		hipDeviceSynchronize();
		randomMagnitude /= 2.0f;
		chunkSize /= 2;
		half = chunkSize / 2;
		algoStep++;
		TwoKElements *= 2;
	}
	hipMemcpy(HeightMap, HeightMapGPU, byteSize, hipMemcpyDeviceToHost);
	//Free memories
	hipFree(states);
	hipFree(HeightMapGPU);
}

void DiamondSquarePAR::PrintMap()
{
	for (int i = 0; i < HeightMapSize; i++)
	{
		for (int j = 0; j < HeightMapSize; j++)
		{
			if (HeightMap[i * HeightMapSize + j] != 0.0f)
			{
				std::cout << HeightMap[i * HeightMapSize + j] << " ";
				continue;
			}
			std::cout << "0" << " ";
		}
		std::cout << "\n";
	}
}





